#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************

  Compile and run with:
    nvcc -o cudapassword cudapassword.cu
    ./cudapassword

*****************************************************************************/



__device__ int is_a_match(char *attempt) {
	char mypassword1[] = "CR2817";
	char mypassword2[] = "LM3019";
	char mypassword3[] = "LS0909";
	char mypassword4[] = "NJ1110";


	char *s = attempt;
	char *h = attempt;
	char *u = attempt;
	char *b = attempt;
	char *pw1 = mypassword1;
	char *pw2 = mypassword2;
	char *pw3 = mypassword3;
	char *pw4 = mypassword4;

	while(*s == *pw1) { 
		if(*s == '\0') 
		{
			printf("Password: %s\n",mypassword1);
			break;
		}

		s++;
		pw1++;
	}
	
	while(*h == *pw2) { 
		if(*h == '\0') 
		{
			printf("Password: %s\n",mypassword2);
			break;
		}

		h++;
		pw2++;
	}

	while(*u == *pw3) { 
		if(*u == '\0') 
		{
			printf("Password: %s\n",mypassword3);
			break;
		}

		u++;
		pw3++;
	}

	while(*b == *pw4) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",mypassword4);
			return 1;
		}

		b++;
		pw4++;
	}
	return 0;
}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


